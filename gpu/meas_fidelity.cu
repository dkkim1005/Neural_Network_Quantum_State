// Copyright (c) 2020 Dongkyu Kim (dkkim1005@gmail.com)

#include <iostream>
#include <iomanip>
#include <memory>
#include <string>
#include <fstream>
#include "meas.cuh"
#include "../cpu/argparse.hpp"

template <typename FloatType> std::string convert_from_float_to_string(const FloatType & num);
template <typename FloatType> std::string path_to_file(const argsparse & parser, const int i);

int main(int argc, char* argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("Ni", "# of input nodes"));
  options.push_back(pair_t("Nh1", "# of hidden nodes 1"));
  options.push_back(pair_t("Nh2", "# of hidden nodes 2"));
  options.push_back(pair_t("ver1", "version 1"));
  options.push_back(pair_t("ver2", "version 2"));
  options.push_back(pair_t("ns", "# of spin samples for parallel Monte-Carlo"));
  options.push_back(pair_t("h1", "transverse-field strength 1"));
  options.push_back(pair_t("h2", "transverse-field strength 2"));
  options.push_back(pair_t("ntrials", "# of trials to compute overlap integral"));
  options.push_back(pair_t("nwarm", "# of MCMC steps for warming-up"));
  options.push_back(pair_t("nms", "# of MCMC steps for sampling spins"));
  options.push_back(pair_t("dev", "device number"));
  options.push_back(pair_t("seed", "seed of the parallel random number generator"));
  options.push_back(pair_t("path", "directory to load and save files"));
  options.push_back(pair_t("lattice", "lattice type(=CH,SQ,TRI,CB)"));
  // env; default value
  defaults.push_back(pair_t("nwarm", "100"));
  defaults.push_back(pair_t("nms", "1"));
  defaults.push_back(pair_t("seed", "0"));
  defaults.push_back(pair_t("path", "."));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);
  const int nInputs = parser.find<int>("Ni"),
    nHiddens1 = parser.find<int>("Nh1"),
    nHiddens2 = parser.find<int>("Nh2"),
    nChains = parser.find<int>("ns"),
    ntrials = parser.find<int>("ntrials"),
    nWarmup = parser.find<int>("nwarm"),
    nMonteCarloSteps = parser.find<int>("nms"),
    deviceNumber = parser.find<int>("dev"),
    ver1 = parser.find<int>("ver1"),
    ver2 = parser.find<int>("ver2");
  const float h1 = parser.find<float>("h1"), h2 = parser.find<float>("h2");
  const unsigned long seed = parser.find<unsigned long>("seed");
  const std::string path1 = path_to_file<float>(parser, 1), path2 = path_to_file<float>(parser, 2);

  // print info of the registered args
  parser.print(std::cout);

  // check whether the cuda device is available
  int devicesCount;
  CHECK_ERROR(hipSuccess, hipGetDeviceCount(&devicesCount));
  if (deviceNumber >= devicesCount)
  {
    std::cerr << "# error ---> dev(" << deviceNumber << ") >= # of devices(" << devicesCount << ")" << std::endl;
    exit(1);
  }
  CHECK_ERROR(hipSuccess, hipSetDevice(deviceNumber));

  ComplexFNN<float> m1(nInputs, nHiddens1, nChains), m2(nInputs, nHiddens2, nChains),
    psi1(nInputs, nHiddens1, nChains), psi2(nInputs, nHiddens1, nChains);

  // load parameters: w,a,b
  m1.load(FNNDataType::W1, path1 + "Dw1.dat");
  m1.load(FNNDataType::W2, path1 + "Dw2.dat");
  m1.load(FNNDataType::B1, path1 + "Db1.dat");
  m2.load(FNNDataType::W1, path2 + "Dw1.dat");
  m2.load(FNNDataType::W2, path2 + "Dw2.dat");
  m2.load(FNNDataType::B1, path2 + "Db1.dat");
  m1.copy_to(psi1);
  m2.copy_to(psi2);

  // block size for the block splitting scheme of parallel Monte-Carlo
  const unsigned long nBlocks = static_cast<unsigned long>(ntrials)*
    static_cast<unsigned long>(nMonteCarloSteps)*
    static_cast<unsigned long>(nInputs)*
    static_cast<unsigned long>(nChains);

  // measurements of the overlap integral for the given wave functions
  struct TRAITS { using AnsatzType = ComplexFNN<float>; using FloatType = float; };

  Sampler4SpinHalf<TRAITS> smp1(m1, seed, nBlocks), smp2(m2, seed+987654321ul, nBlocks);

  MeasFidelity<TRAITS> fidelity(smp1, smp2, psi1, psi2);
  float res = fidelity.measure(ntrials, nWarmup, nMonteCarloSteps);
  std::cout << "# |<\\psi_1|\\psi_2>| : " << res << std::endl;

  // record measurements
  const std::string filename = parser.find<>("lattice")+ "-F-Ni" + std::to_string(nInputs) + ".dat";
  std::ofstream wfile;
  if(!std::ifstream(filename).is_open())
  {
    wfile.open(filename);
    wfile << "#   nh1      h1      v1     nh2      h2      v2    seed       F"
          << std::endl;
  }
  else
    wfile.open(filename, std::ofstream::app);
  // format: nh1 h1 v1 nh2 h2 v2 seed F
  wfile << std::setprecision(7);
  wfile << std::setw(7) << nHiddens1 << " "
        << std::setw(7) << h1 << " "
        << std::setw(7) << ver1 << " "
        << std::setw(7) << nHiddens2 << " "
        << std::setw(7) << h2 << " "
        << std::setw(7) << ver2 << " "
        << std::setw(7) << seed << " "
        << std::setw(7) << res << std::endl;
  wfile.close();
  return 0;
}

template <typename FloatType>
std::string convert_from_float_to_string(const FloatType & num)
{
  std::string numstr = std::to_string(num);
  numstr.erase(numstr.find_last_not_of('0') + 1, std::string::npos);
  numstr.erase(numstr.find_last_not_of('.') + 1, std::string::npos);
  return numstr;
}

template <typename FloatType>
std::string path_to_file(const argsparse & parser, const int i)
{
  const std::string hQuery = "h"+std::to_string(i),
    NhQuery = "Nh"+std::to_string(i),
    vQuery = "ver"+std::to_string(i);
  const std::string filepath = parser.find<>("path") + "/"
    + parser.find<>("lattice") + "-Ni" + parser.find<>("Ni") + "Nh"
    + parser.find<>(NhQuery) + "Hf" + convert_from_float_to_string(parser.find<FloatType>(hQuery))
    + "V" + parser.find<>(vQuery);
  return filepath;
}
