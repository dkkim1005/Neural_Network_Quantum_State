// Copyright (c) 2020 Dongkyu Kim (dkkim1005@gmail.com)

#include "measurements.cuh"
#include "hamiltonians.cuh"
#include "../cpu/argparse.hpp"

int main(int argc, char* argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("L", "# of lattice sites"));
  options.push_back(pair_t("nh", "# of hidden nodes"));
  options.push_back(pair_t("ns", "# of spin samples for parallel Monte-Carlo"));
  options.push_back(pair_t("niter", "# of iterations to sample the ground energy"));
  options.push_back(pair_t("h", "transverse-field strength"));
  options.push_back(pair_t("ver", "version"));
  options.push_back(pair_t("nwarm", "# of MCMC steps for warming-up"));
  options.push_back(pair_t("nms", "# of MCMC steps for sampling spins"));
  options.push_back(pair_t("dev", "device number"));
  options.push_back(pair_t("J", "coupling constant"));
  options.push_back(pair_t("path", "directory to load and save files"));
  options.push_back(pair_t("seed", "seed of the parallel random number generator"));
  options.push_back(pair_t("ifprefix", "prefix of the file to load data"));
  options.push_back(pair_t("lattice", "lattice type(=CH,SQ,TRI,CB)"));
  // env; default value
  defaults.push_back(pair_t("nwarm", "100"));
  defaults.push_back(pair_t("nms", "1"));
  defaults.push_back(pair_t("J", "-1.0"));
  defaults.push_back(pair_t("path", "."));
  defaults.push_back(pair_t("seed", "0"));
  defaults.push_back(pair_t("ifprefix", "None"));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);

  const uint32_t L = parser.find<uint32_t>("L"),
    nInputs = L*L,
    nHiddens = parser.find<uint32_t>("nh"),
    nChains = parser.find<uint32_t>("ns"),
    nWarmup = parser.find<uint32_t>("nwarm"),
    nMonteCarloSteps = parser.find<uint32_t>("nms"),
    deviceNumber = parser.find<uint32_t>("dev"),
    nIterations =  parser.find<uint32_t>("niter"),
    version = parser.find<uint32_t>("ver");
  const double h = parser.find<double>("h"), J = parser.find<double>("J");
  const uint64_t seed = parser.find<uint64_t>("seed");
  const std::string path = parser.find<>("path") + "/",
    nistr = std::to_string(nInputs),
    nhstr = std::to_string(nHiddens),
    vestr = std::to_string(version),
    ifprefix = parser.find<>("ifprefix");
  std::string hfstr = std::to_string(h);
  hfstr.erase(hfstr.find_last_not_of('0') + 1, std::string::npos);
  hfstr.erase(hfstr.find_last_not_of('.') + 1, std::string::npos);

  // print info of the registered args
  parser.print(std::cout);

  // check whether the cuda device is available
  int devicesCount;
  CHECK_ERROR(hipSuccess, hipGetDeviceCount(&devicesCount));
  if (deviceNumber >= devicesCount)
  {
    std::cerr << "# error ---> dev(" << deviceNumber << ") >= # of devices(" << devicesCount << ")" << std::endl;
    exit(1);
  }
  CHECK_ERROR(hipSuccess, hipSetDevice(deviceNumber));

  ComplexFNN<double> machine(nInputs, nHiddens, nChains);

  // load parameters
  const std::string lattice = parser.find<>("lattice") + "-",
    prefix = path + lattice + "Ni" + nistr + "Nh" + nhstr + "Hf" + hfstr + "V" + vestr;
  const std::string prefix0 = (ifprefix.compare("None")) ? path+ifprefix : prefix;

  machine.load(FNNDataType::W1, prefix0 + "Dw1.dat");
  machine.load(FNNDataType::W2, prefix0 + "Dw2.dat");
  machine.load(FNNDataType::B1, prefix0 + "Db1.dat");

  struct SamplerTraits { using AnsatzType = ComplexFNN<double>; using FloatType = double;};

  // block size for the block splitting scheme of parallel Monte-Carlo
  const uint64_t nBlocks = static_cast<uint64_t>(nIterations)*
                           static_cast<uint64_t>(nMonteCarloSteps)*
                           static_cast<uint64_t>(nInputs)*
                           static_cast<uint64_t>(nChains);

  // Transverse Field Ising Hamiltonian on the square lattice
  using SamplerType = spinhalf::TFISQ<SamplerTraits>;
  SamplerType sampler(machine, L, h, J, seed, nBlocks);

  const auto start = std::chrono::system_clock::now();
  sampler.warm_up(nWarmup);

  const double groundEnergy = meas_energy<SamplerType, double>(sampler, nIterations, nWarmup, nMonteCarloSteps);

  std::cout << std::setprecision(7) << groundEnergy << std::endl;

  const auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  std::cout << "# elapsed time: " << elapsed_seconds.count() << "(sec)" << std::endl;

  return 0;
}
