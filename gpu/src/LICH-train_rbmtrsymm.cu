#include "hip/hip_runtime.h"
// Copyright (c) 2020 Dongkyu Kim (dkkim1005@gmail.com)

#include "../include/common.cuh"
#include "../include/neural_quantum_state.cuh"
#include "../include/hamiltonians.cuh"
#include "../include/optimizer.cuh"
#include "../../cpu/include/argparse.hpp"

template <typename FloatType>
std::string remove_zeros_in_str(const FloatType val);

using namespace spinhalf;

int main(int argc, char* argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("L", "# of lattice sites"));
  options.push_back(pair_t("nf", "# of filters"));
  options.push_back(pair_t("ns", "# of spin samples for parallel Monte-Carlo"));
  options.push_back(pair_t("niter", "# of iterations to train RBMTrSymm"));
  options.push_back(pair_t("alpha", "exponent in the two-body interaction: J_{i,j} ~ 1/|i-j|^{alpha}"));
  options.push_back(pair_t("theta", "J = sin(theta), h = -cos(theta)"));
  options.push_back(pair_t("ver", "version"));
  options.push_back(pair_t("nwarm", "# of MCMC steps for warming-up"));
  options.push_back(pair_t("nms", "# of MCMC steps for sampling spins"));
  options.push_back(pair_t("dev", "device number"));
  options.push_back(pair_t("lr", "learning_rate"));
  options.push_back(pair_t("path", "directory to load and save files"));
  options.push_back(pair_t("seed", "seed of the parallel random number generator"));
  options.push_back(pair_t("ifprefix", "prefix of the file to load data"));
  // env; default value
  defaults.push_back(pair_t("nwarm", "100"));
  defaults.push_back(pair_t("nms", "1"));
  defaults.push_back(pair_t("lr", "1e-2"));
  defaults.push_back(pair_t("path", "."));
  defaults.push_back(pair_t("seed", "0"));
  defaults.push_back(pair_t("ifprefix", "None"));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);

  const int L = parser.find<int>("L"),
    nChains = parser.find<int>("ns"),
    nWarmup = parser.find<int>("nwarm"),
    nMonteCarloSteps = parser.find<int>("nms"),
    deviceNumber = parser.find<int>("dev"),
    nIterations =  parser.find<int>("niter");
  const double lr = parser.find<double>("lr");
  const unsigned long long seed = parser.find<unsigned long long>("seed");
  const std::string path = parser.find<>("path") + "/",
    Lstr = parser.find<>("L"),
    ifprefix = parser.find<>("ifprefix");
  const auto nFiltersArr = parser.mfind<int>("nf");
  const auto alphaArr = parser.mfind<double>("alpha");
  const auto verArr = parser.mfind<int>("ver");
  const auto thetaArr = parser.mfind<double>("theta");

  // print info of the registered args
  parser.print(std::cout);

  // check whether the cuda device is available
  int devicesCount;
  CHECK_ERROR(hipSuccess, hipGetDeviceCount(&devicesCount));
  if (deviceNumber >= devicesCount)
  {
    std::cerr << "# error ---> dev(" << deviceNumber << ") >= # of devices(" << devicesCount << ")" << std::endl;
    exit(1);
  }
  CHECK_ERROR(hipSuccess, hipSetDevice(deviceNumber));

  struct SamplerTraits { using AnsatzType = RBMTrSymm<double>; using FloatType = double; };

  // block size for the block splitting scheme of parallel Monte-Carlo
  const unsigned long nBlocks = static_cast<unsigned long>(nIterations)*
    static_cast<unsigned long>(nMonteCarloSteps)*
    static_cast<unsigned long>(L)*
    static_cast<unsigned long>(nChains);

  for (const auto & ver : verArr)
    for (const auto & nf : nFiltersArr)
      for (const auto & alpha : alphaArr)
        for (const auto & theta : thetaArr)
        {
          const std::string verstr = std::to_string(ver),
            nfstr = std::to_string(nf),
            alphastr = remove_zeros_in_str(alpha),
            thetastr = remove_zeros_in_str(theta);
          RBMTrSymm<double> machine(L, nf, nChains);
          const double J = std::sin(theta), h = -std::cos(theta);
          // load parameters
          const std::string prefix = path + "RBMTrSymmLICH-L" + Lstr + "NF" + nfstr + "A" + alphastr + "T" + thetastr + "V" + verstr;
          const std::string prefix0 = (ifprefix.compare("None")) ? path+ifprefix : prefix;
          machine.load(prefix0);
          // Transverse Field Ising Hamiltonian with long-range interaction on the 1D chain lattice
          LITFIChain<SamplerTraits> sampler(machine, L, h, J, alpha, true, seed, nBlocks, prefix);
          const auto start = std::chrono::system_clock::now();
          sampler.warm_up(nWarmup);
          StochasticReconfigurationCG<double> iTimePropagator(nChains, machine.get_nVariables());
          iTimePropagator.propagate(sampler, nIterations, nMonteCarloSteps, lr);
          // save parameters
          machine.save(prefix);
          const auto end = std::chrono::system_clock::now();
          std::chrono::duration<double> elapsed_seconds = end-start;
          std::cout << "# elapsed time: " << elapsed_seconds.count() << "(sec)" << std::endl;
        }

  return 0;
}

template <typename FloatType>
std::string remove_zeros_in_str(const FloatType val)
{
  std::string tmp = std::to_string(val);
  tmp.erase(tmp.find_last_not_of('0') + 1, std::string::npos);
  tmp.erase(tmp.find_last_not_of('.') + 1, std::string::npos);
  return tmp;
}
