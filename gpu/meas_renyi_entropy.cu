// Copyright (c) 2020 Dongkyu Kim (dkkim1005@gmail.com)

#include "meas.cuh"
#include "../cpu/argparse.hpp"

int main(int argc, char* argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("L", "# of lattice sites"));
  options.push_back(pair_t("nh", "# of hidden nodes"));
  options.push_back(pair_t("ns", "# of spin samples for parallel Monte-Carlo"));
  options.push_back(pair_t("niter", "# of iterations to measure Renyi entropy"));
  options.push_back(pair_t("h", "transverse-field strength"));
  options.push_back(pair_t("ver", "version"));
  options.push_back(pair_t("nwarm", "# of MCMC steps for warming-up"));
  options.push_back(pair_t("nms", "# of MCMC steps for sampling spins"));
  options.push_back(pair_t("dev", "device number"));
  options.push_back(pair_t("path", "directory to load files"));
  options.push_back(pair_t("seed", "seed of the parallel random number generator"));
  options.push_back(pair_t("ifprefix", "prefix of the file to load data"));
  // env; default value
  defaults.push_back(pair_t("nwarm", "100"));
  defaults.push_back(pair_t("nms", "1"));
  defaults.push_back(pair_t("path", "."));
  defaults.push_back(pair_t("seed", "0"));
  defaults.push_back(pair_t("ifprefix", "None"));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);

  const int L = parser.find<int>("L"),
    nInputs = L,
    nHiddens = parser.find<int>("nh"),
    nChains = parser.find<int>("ns"),
    nWarmup = parser.find<int>("nwarm"),
    nMonteCarloSteps = parser.find<int>("nms"),
    deviceNumber = parser.find<int>("dev"),
    nIterations =  parser.find<int>("niter"),
    version = parser.find<int>("ver");
  const float h = parser.find<float>("h");
  const unsigned long seed = parser.find<unsigned long>("seed");
  const std::string path = parser.find<>("path") + "/",
    nistr = std::to_string(nInputs),
    nhstr = std::to_string(nHiddens),
    vestr = std::to_string(version),
    ifprefix = parser.find<>("ifprefix");
  std::string hfstr = std::to_string(h);
  hfstr.erase(hfstr.find_last_not_of('0') + 1, std::string::npos);
  hfstr.erase(hfstr.find_last_not_of('.') + 1, std::string::npos);

  // print info of the registered args
  parser.print(std::cout);

  // check whether the cuda device is available
  int devicesCount;
  CHECK_ERROR(hipSuccess, hipGetDeviceCount(&devicesCount));
  if (deviceNumber >= devicesCount)
  {
    std::cerr << "# error ---> dev(" << deviceNumber << ") >= # of devices(" << devicesCount << ")" << std::endl;
    exit(1);
  }
  CHECK_ERROR(hipSuccess, hipSetDevice(deviceNumber));

  ComplexFNN<float> psi1(nInputs, nHiddens, nChains), psi2(nInputs, nHiddens, nChains), psi3(nInputs, nHiddens, nChains);

  // load parameters
  const std::string prefix = path + "CH-Ni" + nistr + "Nh" + nhstr + "Hf" + hfstr + "V" + vestr;
  const std::string prefix0 = (ifprefix.compare("None")) ? path+ifprefix : prefix;

  psi1.load(FNNDataType::W1, prefix0 + "Dw1.dat");
  psi1.load(FNNDataType::W2, prefix0 + "Dw2.dat");
  psi1.load(FNNDataType::B1, prefix0 + "Db1.dat");
  psi1.copy_to(psi2);
  psi1.copy_to(psi3);

  struct TRAITS { using AnsatzType = ComplexFNN<float>; using FloatType = float; };

  // block size for the block splitting scheme of parallel Monte-Carlo
  const unsigned long nBlocks = static_cast<unsigned long>(nIterations)*
                                static_cast<unsigned long>(nMonteCarloSteps)*
                                static_cast<unsigned long>(nInputs)*
                                static_cast<unsigned long>(nChains);

  Sampler4SpinHalf<TRAITS> sampler1(psi1, seed, nBlocks), sampler2(psi2, seed+987654321ul, nBlocks);
  MeasRenyiEntropy<TRAITS> S2measure(sampler1, sampler2, psi3);
  S2measure.measure(nInputs/2, nIterations, nMonteCarloSteps, nWarmup);

  return 0;
}
