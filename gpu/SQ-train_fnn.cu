// Copyright (c) 2020 Dongkyu Kim (dkkim1005@gmail.com)

#include "common.cuh"
#include "neural_quantum_state.cuh"
#include "hamiltonians.cuh"
#include "optimizer.cuh"
#include "../cpu/argparse.hpp"

int main(int argc, char* argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("L", "# of lattice sites"));
  options.push_back(pair_t("nh", "# of hidden nodes"));
  options.push_back(pair_t("ns", "# of spin samples for parallel Monte-Carlo"));
  options.push_back(pair_t("niter", "# of iterations to train FNN"));
  options.push_back(pair_t("h", "transverse-field strength"));
  options.push_back(pair_t("ver", "version"));
  options.push_back(pair_t("nwarm", "# of MCMC steps for warming-up"));
  options.push_back(pair_t("nms", "# of MCMC steps for sampling spins"));
  options.push_back(pair_t("dev", "device number"));
  options.push_back(pair_t("J", "coupling constant"));
  options.push_back(pair_t("lr", "learning_rate"));
  options.push_back(pair_t("path", "directory to load and save files"));
  options.push_back(pair_t("seed", "seed of the parallel random number generator"));
  options.push_back(pair_t("ifprefix", "prefix of the file to load data"));
  options.push_back(pair_t("dr", "dropout rate"));
  // env; default value
  defaults.push_back(pair_t("nwarm", "100"));
  defaults.push_back(pair_t("nms", "1"));
  defaults.push_back(pair_t("J", "-1.0"));
  defaults.push_back(pair_t("lr", "5e-3"));
  defaults.push_back(pair_t("path", "."));
  defaults.push_back(pair_t("seed", "0"));
  defaults.push_back(pair_t("ifprefix", "None"));
  defaults.push_back(pair_t("dr", "5e-1"));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);

  const uint32_t L = parser.find<uint32_t>("L"),
    nInputs = L*L,
    nHiddens = parser.find<int>("nh"),
    nChains = parser.find<int>("ns"),
    nWarmup = parser.find<int>("nwarm"),
    nMonteCarloSteps = parser.find<int>("nms"),
    deviceNumber = parser.find<int>("dev"),
    nIterations =  parser.find<int>("niter"),
    version = parser.find<int>("ver");
  const double h = parser.find<double>("h"),
    J = parser.find<double>("J"),
    lr = parser.find<double>("lr"),
    dr = parser.find<double>("dr");
  const uint64_t seed = parser.find<uint64_t>("seed");
  const std::string path = parser.find<>("path") + "/",
    nistr = std::to_string(nInputs),
    nhstr = std::to_string(nHiddens),
    vestr = std::to_string(version),
    ifprefix = parser.find<>("ifprefix");
  std::string hfstr = std::to_string(h);
  hfstr.erase(hfstr.find_last_not_of('0') + 1, std::string::npos);
  hfstr.erase(hfstr.find_last_not_of('.') + 1, std::string::npos);

  // print info of the registered args
  parser.print(std::cout);

  // check whether the cuda device is available
  int devicesCount;
  CHECK_ERROR(hipSuccess, hipGetDeviceCount(&devicesCount));
  if (deviceNumber >= devicesCount)
  {
    std::cerr << "# error ---> dev(" << deviceNumber << ") >= # of devices(" << devicesCount << ")" << std::endl;
    exit(1);
  }
  CHECK_ERROR(hipSuccess, hipSetDevice(deviceNumber));

  ComplexFNN<double> machine(nInputs, nHiddens, nChains);

  // load parameters
  const std::string prefix = path + "SQ-Ni" + nistr + "Nh" + nhstr + "Hf" + hfstr + "V" + vestr;
  const std::string prefix0 = (ifprefix.compare("None")) ? path+ifprefix : prefix;

  machine.load(FNNDataType::W1, prefix0 + "Dw1.dat");
  machine.load(FNNDataType::W2, prefix0 + "Dw2.dat");
  machine.load(FNNDataType::B1, prefix0 + "Db1.dat");

  struct SamplerTraits { using AnsatzType = ComplexFNN<double>; using FloatType = double;};

  // block size for the block splitting scheme of parallel Monte-Carlo
  const uint64_t nBlocks = static_cast<uint64_t>(nIterations)*
                           static_cast<uint64_t>(nMonteCarloSteps)*
                           static_cast<uint64_t>(nInputs)*
                           static_cast<uint64_t>(nChains);

  // Transverse Field Ising Hamiltonian on the square lattice
  spinhalf::TFISQ<SamplerTraits> sampler(machine, L, h, J, seed, nBlocks, dr, prefix);

  const auto start = std::chrono::system_clock::now();

  sampler.warm_up(nWarmup);

  const int nCutHiddens = static_cast<int>(nHiddens*dr);
  const int nVariables = nInputs*nCutHiddens + 2*nCutHiddens;
  StochasticReconfigurationCG<double> iTimePropagator(nChains, nVariables);
  iTimePropagator.propagate(sampler, nIterations, nMonteCarloSteps, lr);

  // save parameters
  machine.save(FNNDataType::W1, prefix + "Dw1.dat");
  machine.save(FNNDataType::W2, prefix + "Dw2.dat");
  machine.save(FNNDataType::B1, prefix + "Db1.dat");

  const auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  std::cout << "# elapsed time: " << elapsed_seconds.count() << "(sec)" << std::endl;

  return 0;
}
